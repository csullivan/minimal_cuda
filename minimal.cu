#include "a_cpu_class.hh"
#include "a_mixed_gpu_and_cpu_class.hh"


int main(int argc, char** argv) {

  hipDeviceSynchronize();
  a_cpu_class cpu_obj;
  a_mixed_gpu_and_cpu_class mixed_obj;

  cpu_obj.set_member(10);
  mixed_obj.a_host_function();

  return 0;

}
